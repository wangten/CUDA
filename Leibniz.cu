
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
__global__ void ken(double *a)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  a[id]=pow((double)(4*id+1),-1)-pow((double)(4*id+3),-1);
}
__global__ void ken2(double *a,double *b,int *dcount)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  int count=*dcount;
  if(count%2==0)
  {
    count=count/2;
  }
  else
  {
    count=count/2+1;
  }
  if(id<count)
  {
    b[id]=a[id*2]+a[id*2+1];
    a[id*2]=0.0;
    a[id*2+1]=0.0;  
  }
  if(id==0)
    *dcount=count;
}
int main()
{
  clock_t t1, t2;
  int block =50000;
  int thread=300;
  int count=block*thread;
  int size=block*thread;
  int *dcount;
  double *a,*da,*db;
  a=(double*)malloc(size*sizeof(double));
  hipMalloc((void**)&da,size*sizeof(double));
  hipMalloc((void**)&db,size*sizeof(double));
  hipMalloc((void**)&dcount,sizeof(int));
 
  t1 = clock();
  hipMemcpy(dcount,&count,sizeof(int),hipMemcpyHostToDevice);
  ken<<<block,thread>>>(da);
  while(count>1)
  {
    if(count%2==0)
      count=count/2;
    else
      count=count/2+1;
    ken2<<<block,thread>>>(da,db,dcount);
    hipDeviceSynchronize();
    hipMemcpy(da,db,size*sizeof(double),hipMemcpyDeviceToDevice);

  }
  hipMemcpy(a,da,size*sizeof(double),hipMemcpyDeviceToHost);
  t2 = clock();
  printf("\na[0]=%.8lf\n size=%d\n",4*a[0],size*2);
  printf("%lf\n", (t2-t1)/(double)(CLOCKS_PER_SEC));
  return 0;
}
